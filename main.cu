#include <hip/hip_runtime.h>
#include <stdio.h>

#define cuda_safe_call(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code,
                      const char *file,
                      int line,
                      bool abort=true)
{
   if (code != hipSuccess)
   {
      fprintf(stderr,"GPUassert: %s %s %d\n",
              hipGetErrorString(code),
              file,
              line);
      if (abort)
        exit(code);
   }
}


__global__ void kernel_hello()
{
    printf("Hello from the device.\n");
}

int main()
{
    printf("Hello from the host.\n");
    kernel_hello<<<1,1>>>();
    cuda_safe_call(hipDeviceSynchronize());
    printf("Success!\n");
    return 0;
}
